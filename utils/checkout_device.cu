#include <iostream>
#include <hip/hip_runtime.h>

int main() {
    int device_count;
    hipError_t error = hipGetDeviceCount(&device_count);

    if (error != hipSuccess) {
        std::cerr << "Error: " << hipGetErrorString(error) << std::endl;
        return -1;
    }

    for (int device = 0; device < device_count; ++device) {
        hipDeviceProp_t deviceProp;
        hipGetDeviceProperties(&deviceProp, device);

        std::cout << "Device " << device << ": " << deviceProp.name << std::endl;
        std::cout << "  Total global memory: " << deviceProp.totalGlobalMem / (1 << 20) << " MB" << std::endl;
        std::cout << "  Shared memory per block: " << deviceProp.sharedMemPerBlock << " bytes" << std::endl;
        std::cout << "  Shared memory per multiprocessor: " << deviceProp.sharedMemPerMultiprocessor << " bytes" << std::endl;
        std::cout << "  Max dynamic shared memory per block: " << deviceProp.sharedMemPerBlockOptin << " bytes" << std::endl;
        std::cout << "  Number of multiprocessors: " << deviceProp.multiProcessorCount << std::endl;
        std::cout << "  Compute capability: " << deviceProp.major << "." << deviceProp.minor << std::endl;
        std::cout << std::endl;
    }

    return 0;
}